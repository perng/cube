#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "dbldbl.h"

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

__global__ void solve_quadratic_eq (double a, double b, double c, double *res)
                              
{
    /* Compute solutions in double precision using standard quadratic formula*/
    res[0] = (-b + sqrt (b*b - 4.0*a*c)) / (2.0 * a);
    res[1] = (-b - sqrt (b*b - 4.0*a*c)) / (2.0 * a);

    /* Compute solutions in double-double using standard quadratic formula */
    dbldbl aa = make_dbldbl (a, 0.0);
    dbldbl bb = make_dbldbl (b, 0.0);
    dbldbl cc = make_dbldbl (c, 0.0);
    dbldbl four = make_dbldbl (4.0, 0.0);

    dbldbl zz = neg_dbldbl (bb);           // -b
    dbldbl yy = mul_double_to_dbldbl (b,b);// b*b
    dbldbl ww = mul_dbldbl (four, aa);     // 4*a
    dbldbl vv = mul_dbldbl (ww, cc);       // 4*a*c
    dbldbl uu = sub_dbldbl (yy, vv);       // b*b - 4*a*c
    dbldbl tt = sqrt_dbldbl (uu);          // sqrt (b*b - 4*a*c)
    dbldbl rr = add_double_to_dbldbl (a,a);// 2*a
    dbldbl qq = add_dbldbl (zz, tt);       // -b + sqrt (b*b - 4*a*c)
    dbldbl pp = sub_dbldbl (zz, tt);       // -b - sqrt (b*b - 4*a*c)
    dbldbl xx1 = div_dbldbl (qq, rr);      // (-b + sqrt (b*b - 4*a*c)) / (2*a)
    dbldbl xx2 = div_dbldbl (pp, rr);      // (-b - sqrt (b*b - 4*a*c)) / (2*a)

    res[2] = get_dbldbl_head(xx1) + get_dbldbl_tail(xx1);
    res[3] = get_dbldbl_head(xx2) + get_dbldbl_tail(xx2);

    /* Compute solutions in double precision using more robust formula */
    double q = -0.5 * (b + copysign (sqrt (b*b - 4.0*a*c), b));
    res[4] = q / a;
    res[5] = c / q;
}

int main (void)
{
    /* Naive computation of the solution of a quadratic equation using both 
       double precision and double-double computation, using an example from
       George E. Forsythe, How Do You Solve a Quadratic Equation, Technical
       Report No. CS40, Computer Science Department, Stanford University, 
       June 1966.
    */
    double a = 1.0;
    double b = -100000.0;
    double c = 1.0;
    double *res = 0;
    double x1d, x2d, x1dd, x2dd, x1r, x2r;

    printf ("\nSolving quadratic equation with a = %g  b = %g  c = %g\n", 
            a, b, c);

    CUDA_SAFE_CALL (hipMalloc ((void**)&res, 6*sizeof(double)));
    solve_quadratic_eq<<<1,1>>>(a, b, c, res);
    CUDA_SAFE_CALL (hipDeviceSynchronize());
    CUDA_SAFE_CALL (hipMemcpy (&x1d, &res[0], sizeof(x1d),
                                hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL (hipMemcpy (&x2d, &res[1], sizeof(x2d),
                                hipMemcpyDeviceToHost));   
    CUDA_SAFE_CALL (hipMemcpy (&x1dd, &res[2], sizeof(x1dd),
                                hipMemcpyDeviceToHost));   
    CUDA_SAFE_CALL (hipMemcpy (&x2dd, &res[3], sizeof(x2dd),
                                hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL (hipMemcpy (&x1r, &res[4], sizeof(x1r),
                                hipMemcpyDeviceToHost));   
    CUDA_SAFE_CALL (hipMemcpy (&x2r, &res[5], sizeof(x2r),
                                hipMemcpyDeviceToHost));

    printf ("\nUsing double precision (std. quadratic formula):\n");
    printf ("x1 =% 18.11e   a*x1**2+b*x1+c =% 18.11e\n",
            x1d, a*x1d*x1d+b*x1d+c);
    printf ("x2 =% 18.11e   a*x2**2+b*x2+c =% 18.11e\n", 
            x2d, a*x2d*x2d+b*x2d+c);

    printf ("\nUsing double-double (std. quadratic formula):\n");
    printf ("x1 =% 18.11e   a*x1**2+b*x1+c =% 18.11e\n", 
            x1dd, a*x1dd*x1dd+b*x1dd+c);
    printf ("x2 =% 18.11e   a*x2**2+b*x2+c =% 18.11e\n",
            x2dd, a*x2dd*x2dd+b*x2dd+c);

    printf ("\nUsing double precision (more robust formula):\n");
    printf ("x1 =% 18.11e   a*x1**2+b*x1+c =% 18.11e\n",
            x1r, a*x1r*x1r+b*x1r+c);
    printf ("x2 =% 18.11e   a*x2**2+b*x2+c =% 18.11e\n", 
            x2r, a*x2r*x2r+b*x2r+c);

    CUDA_SAFE_CALL (hipFree(res));
    
    return EXIT_SUCCESS;
}
